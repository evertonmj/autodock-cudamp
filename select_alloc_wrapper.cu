#include "hip/hip_runtime.h"
/*
 * Wrapper for selection allocation
 * Compiled with Cuda compiler.
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "cutil.h"
#include <stdio.h>
#include "typedefs.h"
#include "select_alloc_wrapper.h"

#define BLOCK_SIZE 128

/**
 * GPU kernel for alloc
 * @param alloc_data alloc array to be stored into
 */
__global__ void alloc_kernel(float *alloc_data)
{
    int idx = blockIdx.x  * blockDim.x + threadIdx.x;

//    alloc_data[idx] = make_float4(1.0,1.0,1.0,1.0);
    alloc_data[idx] = 1.0;

}

////////////////////////////////////////////////////////////////////////////////
//! Entry point for Cuda function
//! @param num_individuals number of individuals in population
//! @param alloc CPU alloc array
//! @param bratwurst worst value
//! @param energy
//! @param invdiffwa
////////////////////////////////////////////////////////////////////////////////
//extern "C" void
extern "C" void select_alloc_wrapper(unsigned int num_individuals, Real *alloc, double bratwurst, double energy, double indvwa)
{
    const unsigned int mem_size = sizeof(float) * num_individuals;
    hipError_t kerr;
    int nBlocks;

    // allocate device memory
    float* alloc_data;
    hipMalloc((void**) &alloc_data, mem_size);

    // setup execution parameters
    nBlocks = num_individuals/BLOCK_SIZE + ((num_individuals%BLOCK_SIZE==0)?0:1);

    // execute the kernel
    alloc_kernel<<< nBlocks, BLOCK_SIZE >>>((float *) alloc_data);
    kerr = hipGetLastError();

    if (kerr != hipSuccess)
    { fprintf(stderr, "CUDA ERROR = %s\n", hipGetErrorString(kerr)); }

    // copy results from device to host
    hipMemcpy(alloc, alloc_data, mem_size, hipMemcpyDeviceToHost);

    // cleanup memory
    hipFree(alloc_data);
}
