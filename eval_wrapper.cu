#include "hip/hip_runtime.h"
/*
 * Wrapper for selection allocation
 * Compiled with Cuda compiler.
 */

// includes, system
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "typedefs.h"

//C++ #defines
#include "autocomm.h"
#include "grid.h"
#include "eval.h"
#include "constants.h"
#include "trilinterp.h"
#include "eintcal.h"
#include "distdepdiel.h"
#include "cuda_wrapper.h"
#include "eval_wrapper.h"

texture<float, 1> tex;

// Variables required for energy calculations
//added extern to variables. Everton Mendonça 03/04/2016
int *B_outsidesgpu;
float *energiesgpu;
unsigned int *evalflagsgpu;
int cpunatoms;
int nBlocks;
int blocksize;
unsigned int num_individuals;
float *nb_group_energycpu;

float *float_arraygpu;
int *int_arraygpu;

//tril params
float *crdsgpu;
float *chargesgpu;
float *ABSchargesgpu;
int   *typesgpu;
float *gridinfosgpu;
int   *ignore_intersgpu;
float *cudamap;

//eint cal params
float *nonbondlistsgpu;
Boole *incelecgpu;
Boole inc14interactgpu;
float *scale14sgpu;
Boole *usenonbondcutsgpu;
Boole haveflexresiduesgpu;
float *unboundinternalFEsgpu;
float *evdWHbgpu;
float *solfngpu;
float *epsilonfngpu;
float *repsilonfngpu;
int *nnb_arraygpu;
float *nb_group_energygpu;

#define BLOCK_SIZE 128

// Global variables required for energy calculations
/*int ElecMap = 0;
int DesolvMap = 0;
Real nb_group_energy[3];
int Nnb_array[3];

void CHECK_ERROR(int num);

/**
 * Trilinterp GPU kernel, does trilinterp energy calculations for each
 * individual in the population.
 * @param num_individualsgpu number of individuals in population
 * @param penergiesgpu array of energies used to store individual's energy
 * @param b_comp_intermolgpu flag (used in cpu trilinterp)
 * @param natomsgpu number of atoms (used in cpu trilinterp)
 * @param crdsgpu (used in cpu trilinterp)
 * @param chargesgpu (used in cpu trilinterp)
 * @param ABSchargesgpu (used in cpu trilinterp)
 * @param typesgpu (used in cpu trilinterp)
 * @param ignore_intersgpu (used in cpu trilinterp)
 * @param p_elec_total (used in cpu trilinterp)
 * @param p_emap_total (used in cpu trilinterp)
 * @param elecMap (used in cpu trilinterp)
 * @param desolvMap (used in cpu trilinterp)
 * @param SomeAtomsOutside (used in cpu trilinterp)
 * @param AllAtomsInside (used in cpu trilinterp)
 * @param pfloat_arraygpu array of float variables used in cpu trilinterp
 * @param pint_arraygpu array of integer varibales used in cpu trilinterp
 */

__global__ void eval_tril_kernel(unsigned int num_individualsgpu,
                                float *penergiesgpu,
                                Boole b_comp_intermolgpu,
                                int natomsgpu,
                                float *crdsgpu,
                   float *chargesgpu,
                   float *ABSchargesgpu,
                   int *typesgpu,
                   int * ignore_intersgpu,
                   float *p_elec_total,
                   float *p_emap_total,
                   int elecMap,
                   int desolvMap,
                   int SomeAtomsOutside,
                   int AllAtomsInside,
                   float *pfloat_arraygpu,
                   int *pint_arraygpu)
{
    int idx = blockIdx.x  * blockDim.x + threadIdx.x;

    if (idx < num_individualsgpu)
    {
        int some_atoms_outside_grid;

        if ((int)pint_arraygpu[INTBOUTS * num_individualsgpu + idx]) //B_outsidesgpu[idx])
        {
            some_atoms_outside_grid = SomeAtomsOutside;
        }
        else
        {
            some_atoms_outside_grid = AllAtomsInside;
        }

        if (!(unsigned int)pint_arraygpu[INTEVALFLAG * num_individualsgpu + idx])//!evalflagsgpu[idx])
        {
            if (b_comp_intermolgpu)
            {
//            fprintf(stderr, "woot\n");
                float elec_total=0.0f, emap_total=0.0f;
                int i;

                for (i=0; i<natomsgpu;i++)
                {
                    float e, m, d;
                    float u, v, w;
                    float p0u, p0v, p0w;
                    float p1u, p1v, p1w;
                    int AtomType;
                    int u0, v0, w0;
                    int u1, v1, w1;
                    if (ignore_intersgpu[i])
                    {
                        //if (elec != NULL) elec[i] = 0;
                        //if (emap != NULL) emap[i] = 0;
                        continue;
                    }
                    if (some_atoms_outside_grid)
                    {
                        float x,y,z;
                        x = crdsgpu[idx * natomsgpu * SPACE + i * SPACE + X];
                        y = crdsgpu[idx * natomsgpu * SPACE + i * SPACE + Y];
                        z = crdsgpu[idx * natomsgpu * SPACE + i * SPACE + Z];

                        if (((x)<=((float)pfloat_arraygpu[FLOATINFO * num_individualsgpu + 3])) || ((x)>=((float)pfloat_arraygpu[FLOATINFO * num_individualsgpu + 7])) || ((y)<=((float)pfloat_arraygpu[FLOATINFO * num_individualsgpu + 4])) ||
                                ((y)>=((float)pfloat_arraygpu[FLOATINFO * num_individualsgpu + 8])) || ((z)<=((float)pfloat_arraygpu[FLOATINFO * num_individualsgpu + 5])) || ((z)>=((float)pfloat_arraygpu[FLOATINFO * num_individualsgpu + 9])))
                        {
                            float epenalty;
                            x -= (float)pfloat_arraygpu[FLOATINFO * num_individualsgpu + 0];
                            y -= (float)pfloat_arraygpu[FLOATINFO * num_individualsgpu + 1];
                            z -= (float)pfloat_arraygpu[FLOATINFO * num_individualsgpu + 2];

                            epenalty = sqhypotenuse(x,y,z) * ENERGYPENALTY;
                            //if (elec != NULL) elec[i] = epenalty;
                            //if (emap != NULL) emap[i] = epenalty;
                            elec_total += epenalty;
                            emap_total += epenalty;
                            continue;
                        }
                    }


                    AtomType = typesgpu[i];

                    u1  = (u0 = (int) (u = ((float)crdsgpu[idx * natomsgpu * SPACE + i * SPACE + X]-(float)pfloat_arraygpu[FLOATINFO * num_individualsgpu + 3]) * (float)pfloat_arraygpu[FLOATINFO * num_individualsgpu + 6])) + 1;
                    p1u = 1.0f - (p0u = u - (float) u0);

                    v1  = (v0 = (int) (v = ((float)crdsgpu[idx * natomsgpu * SPACE + i * SPACE + Y]-(float)pfloat_arraygpu[FLOATINFO * num_individualsgpu + 4]) * (float)pfloat_arraygpu[FLOATINFO * num_individualsgpu + 6])) + 1;
                    p1v = 1.0f - (p0v = v - (float) v0);

                    w1  = (w0 = (int) (w = ((float)crdsgpu[idx * natomsgpu * SPACE + i * SPACE + Z]-(float)pfloat_arraygpu[FLOATINFO * num_individualsgpu + 5]) * (float)pfloat_arraygpu[FLOATINFO * num_individualsgpu + 6])) + 1;
                    p1w = 1.0f - (p0w = w - (float) w0);


            #ifdef MINPOINT
                    int ix,iy,iz;                      //MINPOINT
                    ix = (p0u < p1u)? u0 : u1;                  //MINPOINT
                    iy = (p0v < p1v)? v0 : v1;                  //MINPOINT
                    iz = (p0w < p1w)? w0 : w1;                  //MINPOINT


                    e = tex1Dfetch(tex, iz * MAX_GRID_PTS * MAX_GRID_PTS * MAX_MAPS + iy * MAX_GRID_PTS * MAX_MAPS + ix * MAX_MAPS + elecMap);               //MINPOINT
                    m = tex1Dfetch(tex, iz * MAX_GRID_PTS * MAX_GRID_PTS * MAX_MAPS + iy * MAX_GRID_PTS * MAX_MAPS + ix * MAX_MAPS + AtomType);              //MINPOINT
                    d = tex1Dfetch(tex, iz * MAX_GRID_PTS * MAX_GRID_PTS * MAX_MAPS + iy * MAX_GRID_PTS * MAX_MAPS + ix * MAX_MAPS + desolvMap);             //MINPOINT
            #else

                    e = m = d = 0.0f;

                    e += p1u * p1v * p1w * tex1Dfetch(tex, w0 * MAX_GRID_PTS * MAX_GRID_PTS * MAX_MAPS + v0 * MAX_GRID_PTS * MAX_MAPS + u0 * MAX_MAPS + elecMap);
                    m += p1u * p1v * p1w * tex1Dfetch(tex, w0 * MAX_GRID_PTS * MAX_GRID_PTS * MAX_MAPS + v0 * MAX_GRID_PTS * MAX_MAPS + u0 * MAX_MAPS + AtomType);
                    d += p1u * p1v * p1w * tex1Dfetch(tex, w0 * MAX_GRID_PTS * MAX_GRID_PTS * MAX_MAPS + v0 * MAX_GRID_PTS * MAX_MAPS + u0 * MAX_MAPS + desolvMap);

                    d += p0u * p1v * p1w * tex1Dfetch(tex, w0 * MAX_GRID_PTS * MAX_GRID_PTS * MAX_MAPS + v0 * MAX_GRID_PTS * MAX_MAPS + u1 * MAX_MAPS + desolvMap);
                    m += p0u * p1v * p1w * tex1Dfetch(tex, w0 * MAX_GRID_PTS * MAX_GRID_PTS * MAX_MAPS + v0 * MAX_GRID_PTS * MAX_MAPS + u1 * MAX_MAPS + AtomType);
                    e += p0u * p1v * p1w * tex1Dfetch(tex, w0 * MAX_GRID_PTS * MAX_GRID_PTS * MAX_MAPS + v0 * MAX_GRID_PTS * MAX_MAPS + u1 * MAX_MAPS + elecMap);

                    e += p1u * p0v * p1w * tex1Dfetch(tex, w0 * MAX_GRID_PTS * MAX_GRID_PTS * MAX_MAPS + v1 * MAX_GRID_PTS * MAX_MAPS + u0 * MAX_MAPS + elecMap);
                    m += p1u * p0v * p1w * tex1Dfetch(tex, w0 * MAX_GRID_PTS * MAX_GRID_PTS * MAX_MAPS + v1 * MAX_GRID_PTS * MAX_MAPS + u0 * MAX_MAPS + AtomType);
                    d += p1u * p0v * p1w * tex1Dfetch(tex, w0 * MAX_GRID_PTS * MAX_GRID_PTS * MAX_MAPS + v1 * MAX_GRID_PTS * MAX_MAPS + u0 * MAX_MAPS + desolvMap);

                    d += p0u * p0v * p1w * tex1Dfetch(tex, w0 * MAX_GRID_PTS * MAX_GRID_PTS * MAX_MAPS + v1 * MAX_GRID_PTS * MAX_MAPS + u1 * MAX_MAPS + desolvMap);
                    m += p0u * p0v * p1w * tex1Dfetch(tex, w0 * MAX_GRID_PTS * MAX_GRID_PTS * MAX_MAPS + v1 * MAX_GRID_PTS * MAX_MAPS + u1 * MAX_MAPS + AtomType);
                    e += p0u * p0v * p1w * tex1Dfetch(tex, w0 * MAX_GRID_PTS * MAX_GRID_PTS * MAX_MAPS + v1 * MAX_GRID_PTS * MAX_MAPS + u1 * MAX_MAPS + elecMap);

                    e += p1u * p1v * p0w * tex1Dfetch(tex, w1 * MAX_GRID_PTS * MAX_GRID_PTS * MAX_MAPS + v0 * MAX_GRID_PTS * MAX_MAPS + u0 * MAX_MAPS + elecMap);
                    m += p1u * p1v * p0w * tex1Dfetch(tex, w1 * MAX_GRID_PTS * MAX_GRID_PTS * MAX_MAPS + v0 * MAX_GRID_PTS * MAX_MAPS + u0 * MAX_MAPS + AtomType);
                    d += p1u * p1v * p0w * tex1Dfetch(tex, w1 * MAX_GRID_PTS * MAX_GRID_PTS * MAX_MAPS + v0 * MAX_GRID_PTS * MAX_MAPS + u0 * MAX_MAPS + desolvMap);

                    d += p0u * p1v * p0w * tex1Dfetch(tex, w1 * MAX_GRID_PTS * MAX_GRID_PTS * MAX_MAPS + v0 * MAX_GRID_PTS * MAX_MAPS + u1 * MAX_MAPS + desolvMap);
                    m += p0u * p1v * p0w * tex1Dfetch(tex, w1 * MAX_GRID_PTS * MAX_GRID_PTS * MAX_MAPS + v0 * MAX_GRID_PTS * MAX_MAPS + u1 * MAX_MAPS + AtomType);
                    e += p0u * p1v * p0w * tex1Dfetch(tex, w1 * MAX_GRID_PTS * MAX_GRID_PTS * MAX_MAPS + v0 * MAX_GRID_PTS * MAX_MAPS + u1 * MAX_MAPS + elecMap);

                    e += p1u * p0v * p0w * tex1Dfetch(tex, w1 * MAX_GRID_PTS * MAX_GRID_PTS * MAX_MAPS + v1 * MAX_GRID_PTS * MAX_MAPS + u0 * MAX_MAPS + elecMap);
                    m += p1u * p0v * p0w * tex1Dfetch(tex, w1 * MAX_GRID_PTS * MAX_GRID_PTS * MAX_MAPS + v1 * MAX_GRID_PTS * MAX_MAPS + u0 * MAX_MAPS + AtomType);
                    d += p1u * p0v * p0w * tex1Dfetch(tex, w1 * MAX_GRID_PTS * MAX_GRID_PTS * MAX_MAPS + v1 * MAX_GRID_PTS * MAX_MAPS + u0 * MAX_MAPS + desolvMap);

                    d += p0u * p0v * p0w * tex1Dfetch(tex, w1 * MAX_GRID_PTS * MAX_GRID_PTS * MAX_MAPS + v1 * MAX_GRID_PTS * MAX_MAPS + u1 * MAX_MAPS + desolvMap);
                    m += p0u * p0v * p0w * tex1Dfetch(tex, w1 * MAX_GRID_PTS * MAX_GRID_PTS * MAX_MAPS + v1 * MAX_GRID_PTS * MAX_MAPS + u1 * MAX_MAPS + AtomType);
                    e += p0u * p0v * p0w * tex1Dfetch(tex, w1 * MAX_GRID_PTS * MAX_GRID_PTS * MAX_MAPS + v1 * MAX_GRID_PTS * MAX_MAPS + u1 * MAX_MAPS + elecMap);

            #endif // not MINPOINT


                    elec_total += e * chargesgpu[i];
                    emap_total += m + d * ABSchargesgpu[i];

                    //if (elec != NULL)
                    //{
                        //elec[i] = chargesgpu[idx * natomsgpu + i]; // e
                    //}
    //                if (emap != NULL) emap[i] =ABSchargesgpu[idx * natomsgpu + i];// m + d * ABSchargesgpu[idx * natomsgpu + i];



                }

    //            if (p_elec_total != NULL) *p_elec_total = elec_total;
    //            if (p_emap_total != NULL) *p_emap_total = emap_total;
                //energiesgpu[idx] = (float)(elec_total+emap_total);
                penergiesgpu[idx] = (float)(elec_total+emap_total);
//                fprintf(stderr," energy = %f, energy gpu = %f\n", (float)(elec_total +emap_total), pfloat_arraygpu[FLOATENERGIES * num_individualsgpu + idx]);
            }

        }
    }
}

/**
 * eintcal GPU kernel, does eintcal energy calculations for each
 * individual in the population.
 * @param num_individualsgpu number of individuals in population
 * @param natomsgpu number of atoms
 * @param penergiesgpu array of energies used to store individual's energy
 * @param nonbondlist (used in cpu eintcal)
 * @param tcoord (used in cpu eintcal)
 * @param B_include_1_4_interactions (used in cpu eintcal)
 * @param B_have_flexible_residues (used in cpu eintcal)
 * @param nnb_array (used in cpu eintcal)
 * @param Nb_group_energy (used in cpu eintcal)
 * @param stre_vdW_Hb (used in cpu eintcal)
 * @param strsol_fn (used in cpu eintcal)
 * @param strepsilon_fn (used in cpu eintcal)
 * @param strr_epsilon_fn (used in cpu eintcal)
 * @param b_comp_intermolgpu (used in cpu eintcal)
 * @param pfloat_arraygpu array of float variables used in cpu trilinterp
 * @param pint_arraygpu array of integer varibales used in cpu trilinterp
 */
__global__ void eintcal_kernel(
                        unsigned int num_individualsgpu,
                        int natomsgpu,
                        float *penergiesgpu,
                        float *nonbondlist,
                        float *tcoord,
                        Boole B_include_1_4_interactions,
                        Boole B_have_flexible_residues,
                        int *nnb_array,
                        float *Nb_group_energy,
                        float *stre_vdW_Hb,
                        float *strsol_fn,
                        float *strepsilon_fn,
                        float *strr_epsilon_fn,
                        Boole b_comp_intermolgpu,
                        float *pfloat_arraygpu,
                        int *pint_arraygpu)
{

    int idx = blockIdx.x  * blockDim.x + threadIdx.x;

    if (idx < num_individualsgpu)
    {

        if (!pint_arraygpu[INTEVALFLAG * num_individualsgpu + idx])//!evalflagsgpu[idx])
        {

    #ifndef EINTCALPRINT
    #   ifndef NOSQRT
            float r = 0.0f;
//            float nbc = B_use_non_bond_cutoff[idx] ? NBC : 999;
            float nbc = (Boole)pint_arraygpu[INTNONBONDCUT * num_individualsgpu + idx] ? NBC : 999;
    #   else
//            float nbc2 = B_use_non_bond_cutoff[idx] ? NBC2 : 999 * 999;
//            float nbc2 = (Boole)pint_arraygpu[INTNONBONDCUT * num_individualsgpu + idx] ? NBC2 : 999 * 999;
            float nbc = (Boole)pint_arraygpu[INTNONBONDCUT * num_individualsgpu + idx] ? NBC2 : 999 * 999;
    #   endif

    #else
    #   ifndef NOSQRT
            float d = 0.0f;
//            float nbc = B_use_non_bond_cutoff[idx] ? NBC : 999;
            float nbc = (Boole)pint_arraygpu[INTNONBONDCUT * num_individualsgpu + idx] ? NBC : 999;
    #   else
//            float nbc2 = B_use_non_bond_cutoff[idx] ? NBC2 : 999 * 999;
            float nbc = (Boole)pint_arraygpu[INTNONBONDCUT * num_individualsgpu + idx] ? NBC2 : 999 * 999;
    #   endif
    #endif

            float dx = 0.0f, dy = 0.0f, dz = 0.0f;
            float r2 = 0.0f;

            float total_e_internal = 0.0f;

            float e_elec = 0.0f;

    #ifdef EINTCALPRINT
            float total_e_elec = 0.0f;
            float total_e_vdW_Hb = 0.0f;
            float e_vdW_Hb = 0.0f;
            float total_e_desolv = 0.0f;
    #endif

            int inb = 0;
            int a1 = 0, a2 = 0;
            int t1 = 0, t2 = 0;
            int nonbond_type = 0;

            int index_1t_NEINT = 0;
            int index_1t_NDIEL = 0;
            int nb_group = 0;
            int inb_from = 0;
            int inb_to = 0;
            int nb_group_max = 1;

            if (B_have_flexible_residues)
            {
                nb_group_max = 3;
            }

            for (nb_group = 0; nb_group < nb_group_max; nb_group++)
            {
    #ifdef EINTCALPRINT
                if (nb_group ==0)
                {
                    //prints stuff
                }
                if (nb_group == 1)
                {
                    //prints stuff
                }
                if (nb_group == 2)
                {
                    //prints stuff
                }
                if ((Boole)pint_arraygpu[INTINCELEC * num_individualsgpu + idx])//B_calcIntElec[idx])
                {
                    //prints stuff
                } else {
                    //prints stuff
                }
    #endif


                if (nb_group == 0)
                {
                    inb_from = 0;
                } else {
                    inb_from = nnb_array[nb_group-1];
                }
                inb_to = nnb_array[nb_group];

                for (inb = inb_from; inb < inb_to; inb++)
                {

                    float e_internal = 0.0f;
                    float e_desolv = 0.0f;

                    a1 = (int)nonbondlist[inb * 7 + 0];
                    a2 = (int)nonbondlist[inb * 7 + 1];
                    t1 = (int)nonbondlist[inb * 7 + 2];
                    t2 = (int)nonbondlist[inb * 7 + 3];

                    nonbond_type = (int)nonbondlist[inb * 7 + 4];
                    float nb_desolv = nonbondlist[inb * 7  + 5];
                    float q1q2 = nonbondlist[inb * 7 + 6];


                    dx = tcoord[idx * natomsgpu * SPACE + a1 * SPACE + X] - tcoord[idx * natomsgpu * SPACE + a2 * SPACE + X];
                    dy = tcoord[idx * natomsgpu * SPACE + a1 * SPACE + Y] - tcoord[idx * natomsgpu * SPACE + a2 * SPACE + Y];
                    dz = tcoord[idx * natomsgpu * SPACE + a1 * SPACE + Z] - tcoord[idx * natomsgpu * SPACE + a2 * SPACE + Z];

    #ifndef NOSQRT
                    r = clamp(hypotenuse(dx,dy,dz), RMIN_ELEC);
                    r2 = r*r;
                    int index = Ang_to_index(r);

    #else
                    r2 = sqhypotenuse(dx,dy,dz);
                    r2 = clamp(r2, RMIN_ELEC2);
                    int index = SqAng_to_index(r2);
    #endif

                    index_1t_NEINT = BoundedNeint(index);
                    index_1t_NDIEL = BoundedNdiel(index);

                    if ((Boole)pint_arraygpu[INTINCELEC * num_individualsgpu + idx])//B_calcIntElec[idx])
                    {
                        float r_dielectric = strr_epsilon_fn[index_1t_NDIEL];
                        e_elec = q1q2 * r_dielectric;
                        e_internal = e_elec;

                    }

                    //if (r2 < nbc2)
                    if (r2 < nbc)
                    {
                        e_desolv = strsol_fn[index_1t_NEINT] * nb_desolv;
                        int myidx;
                        if (B_include_1_4_interactions != 0 && nonbond_type == 4)
                        {
                            myidx = index_1t_NEINT * ATOM_MAPS * ATOM_MAPS + t2 * ATOM_MAPS + t1;
                            if (myidx == NEINT * ATOM_MAPS * ATOM_MAPS)
                            {
//                                e_internal += scale_1_4[idx] * (stre_vdW_Hb[myidx-1] + e_desolv);
                                e_internal += pfloat_arraygpu[FLOATSCALE14 * num_individualsgpu + idx] * (stre_vdW_Hb[myidx-1] + e_desolv);
                            }
                            else
                            {
//                                e_internal += scale_1_4[idx] * (stre_vdW_Hb[myidx] + e_desolv);
                                e_internal += pfloat_arraygpu[FLOATSCALE14 * num_individualsgpu + idx] * (stre_vdW_Hb[myidx] + e_desolv);
                            }
                        } else {
//                            fprintf(stderr," stre_vdW_Hb[%d][%d][%d] = %f\n", index_1t_NEINT, t2, t1, stre_vdW_Hb[index_1t_NEINT * ATOM_MAPS * ATOM_MAPS + t2 * ATOM_MAPS + t1]);
//i                            e_internal += stre_vdW_Hb[index_1t_NEINT * ATOM_MAPS * ATOM_MAPS + t2 * ATOM_MAPS + t1] + e_desolv;
                            myidx = index_1t_NEINT * ATOM_MAPS * ATOM_MAPS + t2 * ATOM_MAPS + t1;
                            if (myidx == NEINT * ATOM_MAPS * ATOM_MAPS)
                            {
                                e_internal += stre_vdW_Hb[myidx-1] + e_desolv;
//                                fprintf(stderr,"NEINT = %d, index = %d, t2 = %d, t1 = %d\n", NEINT, index_1t_NEINT, t2, t1);

                            }
                            else
                            {
                                e_internal += stre_vdW_Hb[myidx] + e_desolv;
                            }

                        }



                    }
                    total_e_internal += e_internal;

    #ifdef EINTCALPRINT
            total_e_desolv  += e_desolv;
            total_e_elec    += e_elec;
            float dielectric = strepsilon_fn[index_1t_NDIEL];

            if ((Boole)pint_arraygpu[INTINCELEC * num_individualsgpu + idx])//B_calcIntElec[idx])
            {
                e_vdW_Hb = e_internal - e_desolv - e_elec;
                // print stuff
            } else {
                e_vdW_Hb = e_internal - e_desolv;
                // print stuff
            }

            total_e_vdW_Hb += e_vdW_Hb;

    #endif

                }

                if (nb_group == INTRA_LIGAND)
                {
                    Nb_group_energy[INTRA_LIGAND] = total_e_internal;
                } else if (nb_group == INTER) {
                    Nb_group_energy[INTER] = total_e_internal - Nb_group_energy[INTRA_LIGAND];
                } else if (nb_group == INTRA_RECEPTOR) {
                    Nb_group_energy[INTRA_RECEPTOR] = total_e_internal - Nb_group_energy[INTRA_LIGAND] - Nb_group_energy[INTER];
                }

            }

    #ifdef EINTCALPRINT
            if((Boole)pint_arraygpu[INTINCELEC * num_individualsgpu + idx])//B_calcIntElec[idx])
            {
                //print stuff
            } else {
                //print stuff
            }
            //print stuff
    #endif
            if(b_comp_intermolgpu)
            {
                //energiesgpu[idx] += ((float)total_e_internal - (float)unboundinternalFEs[idx]);
                penergiesgpu[idx] += ((float)total_e_internal - pfloat_arraygpu[FLOATUNBOUNDINTERNAL * num_individualsgpu + idx]);//(float)unboundinternalFEs[idx]);
            }
            else
            {
                //energiesgpu[idx] = ((float)total_e_internal - (float)unboundinternalFEs[idx]);
                penergiesgpu[idx] = ((float)total_e_internal - pfloat_arraygpu[FLOATUNBOUNDINTERNAL * num_individualsgpu + idx]);//(float)unboundinternalFEs[idx]);
            }

        }

    }

}


////////////////////////////////////////////////////////////////////////////////
//! Entry point for Cuda function
//! @param b_comp_intermol
//! @param crds
//! @param energiescpu array of energies for returning to cpu
//! @param float_arraycpu
//! @param int_arraycpu
////////////////////////////////////////////////////////////////////////////////
extern "C" void eval_wrapper(
                       Boole b_comp_intermol,
                       float *crds,
                       float *energiescpu,
                       float *float_arraycpu,
                       int *int_arraycpu
                        )

{
    hipMemcpy(crdsgpu, crds, sizeof(float) * cpunatoms * SPACE * num_individuals, hipMemcpyHostToDevice);
    CHECK_ERROR(19);

    hipMemcpy(int_arraygpu, int_arraycpu, sizeof(int) * INTSIZE * num_individuals, hipMemcpyHostToDevice);
    CHECK_ERROR(18);
    hipMemcpy(float_arraygpu, float_arraycpu, sizeof(float) * FLOATSIZE * num_individuals, hipMemcpyHostToDevice);
    CHECK_ERROR(17);

    // execute trilinterp kernel
    eval_tril_kernel<<< nBlocks, blocksize >>>(num_individuals,
                energiesgpu,
                b_comp_intermol, cpunatoms,
                crdsgpu, chargesgpu, ABSchargesgpu, typesgpu,
                ignore_intersgpu,
                (float *)NULL_ELEC_TOTAL,
                (float *)NULL_EVDW_TOTAL,
                ElecMap,
                DesolvMap, SOME_ATOMS_OUTSIDE_GRID,
                ALL_ATOMS_INSIDE_GRID,
                float_arraygpu,
                int_arraygpu);

    CHECK_ERROR(666);
    hipDeviceSynchronize();
    CHECK_ERROR(888);

    // execute eintcal kernel
    eintcal_kernel<<< nBlocks, blocksize >>>(
        num_individuals,
        cpunatoms,
        energiesgpu,
        nonbondlistsgpu,
        crdsgpu,
        inc14interactgpu,
        haveflexresiduesgpu,
        nnb_arraygpu,
        nb_group_energygpu,
        evdWHbgpu,
        solfngpu,
        epsilonfngpu,
        repsilonfngpu,
        b_comp_intermol,
        float_arraygpu,
        int_arraygpu);

    CHECK_ERROR(888);
    hipDeviceSynchronize();
    CHECK_ERROR(23);

    hipMemcpy(energiescpu, energiesgpu, sizeof(float) * num_individuals, hipMemcpyDeviceToHost);
    CHECK_ERROR(33);

}


/**
 * Helper function to check if cuda had an error, and outputs the error
 * @param num error number (user defined)
 */
void CHECK_ERROR(int num)
{
    hipError_t kerr;

    kerr = hipGetLastError();

    if (kerr != hipSuccess)
    {
        fprintf(stderr, "################################\nCUDA ERROR %d = %s\n################################\n", num, hipGetErrorString(kerr));
    }

}

/**
 * Allocates memory on the graphics card and copies consistent variables for later use in the gen alg, also
 * store the map into texture memory for fast access.
 * @param natom number of atoms
 * @param nnum_individuals number of individuals in population
 * @param map the map representation
 * @param nonbondlist nonbonded list
 * @param etbl energie table
 * @param charge charges array
 * @param ABScharge ABScharge array
 * @param type type array
 * @param ignore_inter ingnore_inter array
 * @param inc14intereact inc14interact flag
 * @param haveflexresidues haveflexresidues flag
 */
extern "C" void cuda_alloc_wrapper(int natom, int nnum_individuals, maptype map, NonbondParam * nonbondlist, EnergyTables *etbl, Real *charge, Real *ABScharge, int *type, int *ignore_inter, Boole inc14interact, Boole haveflexresidues, Real *entable_solfn, Real *entable_epsilon_fn, Real *entable_r_epsilon_fn,Real e_vdW_Hb[NEINT][ATOM_MAPS][ATOM_MAPS])
{
    int memsize = 0;
    int i,j,k,l;

    cpunatoms = natom;
    num_individuals = nnum_individuals;

    nb_group_energycpu = (float *)malloc(sizeof(float) * 3);
    float *cudamapcpu = (float *)malloc(sizeof(float) *MAX_GRID_PTS * MAX_GRID_PTS * MAX_GRID_PTS * MAX_MAPS);
    //printf("estou em cuda_alloc_wrapper 2\n");
    for (i = 0; i < MAX_GRID_PTS; i++)
    {
        //printf("estou no for 1\n");
        for (j = 0; j < MAX_GRID_PTS; j++)
        {
            //printf("estou no for 2\n");
            for (k = 0; k < MAX_GRID_PTS; k++)
            {
                //printf("estou no for 3\n");
                for (l = 0; l < MAX_MAPS; l++)
                {
                    //printf("estou no for 4\n");
                    cudamapcpu[i * MAX_GRID_PTS * MAX_GRID_PTS * MAX_MAPS + j * MAX_GRID_PTS * MAX_MAPS + k * MAX_MAPS + l] = map[i][j][k][l];
                }
            }
        }
    }

    hipMalloc((void**)&cudamap,  sizeof(float) * MAX_GRID_PTS * MAX_GRID_PTS * MAX_GRID_PTS * MAX_MAPS);
    CHECK_ERROR(75);

    memsize += sizeof(float) * MAX_GRID_PTS * MAX_GRID_PTS * MAX_GRID_PTS * MAX_MAPS;

    hipMemcpy(cudamap, cudamapcpu, sizeof(float) * MAX_GRID_PTS * MAX_GRID_PTS * MAX_GRID_PTS * MAX_MAPS, hipMemcpyHostToDevice);
    CHECK_ERROR(76);

    tex.addressMode[0] = hipAddressModeWrap;
    tex.filterMode = hipFilterModePoint;
    tex.normalized = false;
    hipBindTexture(0, tex, cudamap);
    free(cudamapcpu);
    CHECK_ERROR(77);

    hipMalloc((void**)&crdsgpu, sizeof(float) * cpunatoms * SPACE * num_individuals);
    CHECK_ERROR(5);

    memsize+= sizeof(float) * cpunatoms * SPACE * num_individuals;

    //get gpu memory for first kernel
    hipMalloc((void**)&evalflagsgpu,sizeof(unsigned int) * num_individuals);
    CHECK_ERROR(0);
    hipMalloc((void**)&energiesgpu, sizeof(float) * num_individuals);
    CHECK_ERROR(1);

    float *energiescpus = (float *)malloc(num_individuals * sizeof(float));
    bzero(energiescpus, sizeof(float) * num_individuals);

    hipMemcpy(energiesgpu, energiescpus, sizeof(float) * num_individuals, hipMemcpyHostToDevice);

    free(energiescpus);

    memsize += sizeof(unsigned int) * num_individuals + sizeof(float) * num_individuals;

    int nb_group_max = 3;
    int nb_group = 0;

    int largest = 0;
    for (nb_group = 0; nb_group < nb_group_max; nb_group++)
    {
        if (largest < Nnb_array[nb_group])
        {
            largest = Nnb_array[nb_group];
        }
    }

    float *nonbondlists = (float *)malloc(sizeof(float) * 7 * largest);

    for (k = 0; k < largest; k++)
    {
        nonbondlists[k * 7 + 0] = (float)nonbondlist[k].a1;
        nonbondlists[k * 7 + 1] = (float)nonbondlist[k].a2;
        nonbondlists[k * 7 + 2] = (float)nonbondlist[k].t1;
        nonbondlists[k * 7 + 3] = (float)nonbondlist[k].t2;
        nonbondlists[k * 7 + 4] = (float)nonbondlist[k].nonbond_type;
        nonbondlists[k * 7 + 5] = (float)nonbondlist[k].desolv;
        nonbondlists[k * 7 + 6] = (float)nonbondlist[k].q1q2;
    }

    hipMalloc((void**)&nonbondlistsgpu, sizeof(float) * 7 * largest);
    CHECK_ERROR(34);

    memsize+= sizeof(float) * 7 * largest;

    hipMemcpy(nonbondlistsgpu, nonbondlists, sizeof(float) * 7 * largest, hipMemcpyHostToDevice);
    CHECK_ERROR(55);

    free(nonbondlists);

    blocksize = BLOCK_SIZE;
    nBlocks = num_individuals/BLOCK_SIZE + ((num_individuals%BLOCK_SIZE==0)?0:1);
    // tril params
    hipMalloc((void**)&B_outsidesgpu, sizeof(int) * num_individuals);
    CHECK_ERROR(2);
    CHECK_ERROR(3);
    CHECK_ERROR(4);
    hipMalloc((void**)&chargesgpu, sizeof(float) * cpunatoms);
    CHECK_ERROR(6);
    hipMalloc((void**)&ABSchargesgpu, sizeof(float) * cpunatoms);
    CHECK_ERROR(7);
    hipMalloc((void**)&typesgpu,sizeof(int) * cpunatoms  * num_individuals);
    CHECK_ERROR(8);
    CHECK_ERROR(9);
    hipMalloc((void**)&gridinfosgpu, sizeof(float) * 10);
    CHECK_ERROR(10);
    hipMalloc((void**)&ignore_intersgpu, sizeof(int) * cpunatoms);
    CHECK_ERROR(11);

    memsize += sizeof(float) * cpunatoms * num_individuals;
    memsize += sizeof(float) * cpunatoms * num_individuals;
    memsize += sizeof(int) * cpunatoms * num_individuals + sizeof(float) * 10 + sizeof(int) * cpunatoms * num_individuals;

    //eintcal params
    hipMalloc((void**)&incelecgpu, sizeof(Boole) * num_individuals);
    CHECK_ERROR(36);
    CHECK_ERROR(37);
    hipMalloc((void**)&scale14sgpu, sizeof(float) * num_individuals);
    CHECK_ERROR(38);
    CHECK_ERROR(39);
    hipMalloc((void**)&usenonbondcutsgpu,sizeof(Boole) * num_individuals);
    CHECK_ERROR(40);
    CHECK_ERROR(41);
    hipMalloc((void**)&unboundinternalFEsgpu,sizeof(float) * num_individuals);
    CHECK_ERROR(42);
    hipMalloc((void**)&nnb_arraygpu, sizeof(int) * 3);
    hipMalloc((void **)&nb_group_energygpu, sizeof(float) * 3);

    memsize += sizeof(Boole) * num_individuals + sizeof(float) * num_individuals + sizeof(Boole) * num_individuals + sizeof(float) * num_individuals + sizeof(int) * 3 + sizeof(float) * 3;

    hipMemcpy(nnb_arraygpu, Nnb_array, sizeof(int) * 3, hipMemcpyHostToDevice);

    float *chargescpu = (float *)malloc(sizeof(float) * cpunatoms);
    float *ABSchargescpu = (float *)malloc(sizeof(float) * cpunatoms);
    for (k = 0; k < cpunatoms; k++)
    {
        chargescpu[k] = (float)charge[k];
        ABSchargescpu[k] = (float)ABScharge[k];
    }

    hipMemcpy(ignore_intersgpu, ignore_inter, sizeof(int) * cpunatoms, hipMemcpyHostToDevice);
    CHECK_ERROR(12);

    hipMemcpy(typesgpu, type, sizeof(int) * cpunatoms, hipMemcpyHostToDevice);
    CHECK_ERROR(20);
    hipMemcpy(chargesgpu, chargescpu, sizeof(float) * cpunatoms, hipMemcpyHostToDevice);
    CHECK_ERROR(16);
    hipMemcpy(ABSchargesgpu, ABSchargescpu, sizeof(float) * cpunatoms, hipMemcpyHostToDevice);
    CHECK_ERROR(15);
    free(chargescpu);
    haveflexresiduesgpu = haveflexresidues;
    inc14interactgpu = inc14interact;
    int m;

    float *evdWHb;
    float *solfn;
    float *epsilonfn;
    float *repsilonfn;

    evdWHb = (float *) malloc(sizeof(float) * NEINT * ATOM_MAPS * ATOM_MAPS);
    solfn = (float *) malloc(sizeof(float) * NEINT);
    epsilonfn = (float *)malloc(sizeof(float) * NDIEL);
    repsilonfn = (float *)malloc(sizeof(float) * NDIEL);

    for (j = 0; j < NEINT; j++)
    {
        //solfn[j] = (float) etbl->sol_fn[j];
        solfn[j] = (float) entable_solfn[j];
        for (k = 0; k < ATOM_MAPS; k++)
        {
            for(m = 0; m < ATOM_MAPS; m++)
            {
                //printf("evdw %d - %d - %d: %f\n\n\n", j, k, m, e_vdW_Hb[j][k][m]);
                //evdWHb[j * ATOM_MAPS * ATOM_MAPS + k * ATOM_MAPS + m] = (float)etbl->e_vdW_Hb[j][k][m];
                evdWHb[j * ATOM_MAPS * ATOM_MAPS + k * ATOM_MAPS + m] = (float) e_vdW_Hb[j][k][m];
//                fprintf(stderr,"evdW_Hb[%d][%d][%d] = %f\n", j,k,m,::evaluate.getETblevdWHb(j,k,m));
            }
        }
    }

    for (k = 0; k < NDIEL; k++)
    {
        // epsilonfn[k] = (float)etbl->epsilon_fn[k];
        // repsilonfn[k] = (float)etbl->r_epsilon_fn[k];
        epsilonfn[k] = (float) entable_epsilon_fn[k];
        repsilonfn[k] = (float) entable_r_epsilon_fn[k];
    }

    hipMalloc((void**)&evdWHbgpu, sizeof(float) * NEINT * ATOM_MAPS * ATOM_MAPS);
    CHECK_ERROR(43);
    hipMalloc((void**)&solfngpu,sizeof(float) * NEINT);
    CHECK_ERROR(44);
    hipMalloc((void**)&epsilonfngpu,sizeof(float) * NDIEL);
    CHECK_ERROR(45);
    hipMalloc((void**)&repsilonfngpu,sizeof(float) *NDIEL);
    CHECK_ERROR(46);

    memsize+= sizeof(float) * NEINT * ATOM_MAPS * ATOM_MAPS + sizeof(float) * NEINT + sizeof(float) * NDIEL + sizeof(float) * NDIEL;

    hipMemcpy(evdWHbgpu, evdWHb, sizeof(float) * NEINT * ATOM_MAPS * ATOM_MAPS, hipMemcpyHostToDevice);
    hipMemcpy(solfngpu, solfn, sizeof(float) * NEINT, hipMemcpyHostToDevice);
    CHECK_ERROR(56);
    hipMemcpy(epsilonfngpu, epsilonfn, sizeof(float) * NDIEL, hipMemcpyHostToDevice);
    CHECK_ERROR(57);
    hipMemcpy(repsilonfngpu, repsilonfn, sizeof(float) * NDIEL, hipMemcpyHostToDevice);
    CHECK_ERROR(58);

    nb_group_energycpu[0] = (float)nb_group_energy[0];
    nb_group_energycpu[1] = (float)nb_group_energy[1];
    nb_group_energycpu[2] = (float)nb_group_energy[2];

    hipMemcpy(nb_group_energygpu, nb_group_energycpu, sizeof(float) * 3, hipMemcpyHostToDevice);

    CHECK_ERROR(54);

    free(evdWHb);
    free(solfn);
    free(epsilonfn);
    free(repsilonfn);

    hipMalloc((void**)&int_arraygpu, sizeof(int) * INTSIZE * num_individuals);
    hipMalloc((void**)&float_arraygpu, sizeof(float) * FLOATSIZE * num_individuals);


}

/**
 * Free's graphics memory
 */
extern "C" void cuda_free_wrapper(void)
{
    hipFree(int_arraygpu);
    hipFree(float_arraygpu);
    //#pragma omp barrier
    //{
      //free(nb_group_energycpu);
    //}
    hipFree(cudamap);
    CHECK_ERROR(78);
    hipFree(crdsgpu);
    CHECK_ERROR(79);

    hipFree(nonbondlistsgpu);
    CHECK_ERROR(59);
    hipFree(evdWHbgpu);
    CHECK_ERROR(68);
    hipFree(solfngpu);
    CHECK_ERROR(69);
    hipFree(epsilonfngpu);
    CHECK_ERROR(70);
    hipFree(repsilonfngpu);
    CHECK_ERROR(71);

    //Free memory for trilinterp params
    hipFree(B_outsidesgpu);
    CHECK_ERROR(24);
    hipFree(chargesgpu);
    CHECK_ERROR(25);
    hipFree(ABSchargesgpu);
    CHECK_ERROR(26);
    hipFree(typesgpu);
    CHECK_ERROR(27);
    CHECK_ERROR(28);
    hipFree(gridinfosgpu);
    CHECK_ERROR(29);
    hipFree(ignore_intersgpu);
    CHECK_ERROR(30);
    CHECK_ERROR(32);

    //free eintcal
    hipFree(incelecgpu);
    CHECK_ERROR(61);
    CHECK_ERROR(62);
    hipFree(scale14sgpu);
    CHECK_ERROR(63);
    CHECK_ERROR(64);
    hipFree(usenonbondcutsgpu);
    CHECK_ERROR(65);
    CHECK_ERROR(66);
    hipFree(unboundinternalFEsgpu);
    CHECK_ERROR(67);
    hipFree(nnb_arraygpu);
    hipFree(nb_group_energygpu);

    //Free others
    hipFree(evalflagsgpu);
    CHECK_ERROR(80);
    hipFree(energiesgpu);
    CHECK_ERROR(81);
}
